#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CUDANewton.cu
 Author      : Jakob Vokac
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <fstream>
#include <vector>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include "model.h"
#include "geometry/vector/vec3d.h"
#include "geometry/curve/cubiccrv.h"
#include "geometry/surface/TopParametric.h"
#include "optimizer/optimizer.h"
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

using namespace std;
/**
 * CUDA kernel that computes reciprocal values for a given vector
 */


__global__ void kernel(vec3d *points, double *distances, int size,
		TopParametric *sur1,
		TopParametric *sur2,
		TopParametric *sur3,
		TopParametric *sur4,
		TopParametric *sur5,
		TopParametric *sur6,
		BottomParametric *sur7,
		BottomParametric *sur8,
		BottomParametric *sur9,
		BottomParametric *sur10,
		BottomParametric *sur11,
		BottomParametric *sur12,
		double divHeight
		){

	optimizer op = optimizer(
			bisection(8),
			quadraticInterpolation(8),
			Newton(1.0),
			sur1,
			sur7,
			vec3d(0,0,0),
			0.00000001,
			20,
			2
	);
	__shared__ TopParametric top1, top2, top3, top4, top5, top6;
	__shared__ BottomParametric bot1, bot2, bot3, bot4, bot5, bot6;

	if(threadIdx.x == 0)
		top1 = *sur1;
	if(threadIdx.x == 1)
		top2 = *sur2;
	if(threadIdx.x == 2)
		top3 = *sur3;
	if(threadIdx.x == 3)
		top4 = *sur4;
	if(threadIdx.x == 4)
		top5 = *sur5;
	if(threadIdx.x == 5)
		top6 = *sur6;
	if(threadIdx.x == 6)
		bot1 = *sur7;
	if(threadIdx.x == 7)
		bot2 = *sur8;
	if(threadIdx.x == 8)
		bot3 = *sur9;
	if(threadIdx.x == 9)
		bot4 = *sur10;
	if(threadIdx.x == 10)
		bot5 = *sur11;
	if(threadIdx.x == 11)
		bot6 = *sur12;

	__syncthreads();
	for(int i = 0; i < size; i+= 1024){
		int arrId = threadIdx.x + blockDim.x * blockIdx.x + i;

		if(arrId >= size)
			break;
		vec3d P = points[arrId];

		op.setTopOrBot(P.z > divHeight);

		if(P.x == 0){
			if(P.y == 0){
				op.setTop(&top1);
				op.setBot(&bot1);
			}
			else if(P.y > 0){
				op.setTop(&top2);
				op.setBot(&bot2);
			}
			else if(P.y < 0){
				op.setTop(&top5);
				op.setBot(&bot5);
			}
		}

		if(P.x > 0 && P.y > 0){
			if(P.y/P.x < sin(M_PI/6)/cos(M_PI/6)){
				op.setTop(&top1);
				op.setBot(&bot1);
			}else if(P.y/P.x < sin(2*M_PI/6)/cos(2*M_PI/6)){
				op.setTop(&top1);
				op.setBot(&bot1);
			}else{
				op.setTop(&top2);
				op.setBot(&bot2);
			}
		}else if(P.x < 0 && P.y < 0){
			if(P.y/-P.x < sin(M_PI/6)/cos(M_PI/6)){
				op.setTop(&top3);
				op.setBot(&bot3);
			}else if(P.y/-P.x < sin(2*M_PI/6)/cos(2*M_PI/6)){
				op.setTop(&top3);
				op.setBot(&bot3);
			}else{
				op.setTop(&top2);
				op.setBot(&bot2);
			}
		}else if(P.x < 0 && P.y < 0){
			if(-P.y/-P.x < sin(M_PI/6)/cos(M_PI/6)){
				op.setTop(&top4);
				op.setBot(&bot4);
			}else if(-P.y/-P.x < sin(2*M_PI/6)/cos(2*M_PI/6)){
				op.setTop(&top4);
				op.setBot(&bot4);
			}else{
				op.setTop(&top5);
				op.setBot(&bot5);
			}
		}else if(P.x > 0 && P.y < 0){
			if(-P.y/P.x < sin(M_PI/6)/cos(M_PI/6)){
				op.setTop(&top6);
				op.setBot(&bot6);
			}else if(-P.y/P.x < sin(2*M_PI/6)/cos(2*M_PI/6)){
				op.setTop(&top6);
				op.setBot(&bot6);
			}else{
				op.setTop(&top5);
				op.setBot(&bot5);
			}
		}
		OptState2D loc = op.optimizeForPoint(P);
//		printf("threadidx.x: %d, blockDim.x: %d, blockIdx.x: %d, i: %d, arrId: %d, dist: %lf\n",threadIdx.x, blockDim.x, blockIdx.x, i, arrId, dist);

		distances[arrId] = loc.dist;
	}
}



//
///**
// * Host function that copies the data and launches the work on GPU
// */
//float *gpuReciprocal(float *data, unsigned size)
//{
//	float *rc = new float[size];
//	float *gpuData;
//
//	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
//	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));
//
//	static const int BLOCK_SIZE = 256;
//	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
//	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);
//
//	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
//	CUDA_CHECK_RETURN(hipFree(gpuData));
//	return rc;
//}
//
//float *cpuReciprocal(float *data, unsigned size)
//{
//	float *rc = new float[size];
//	for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
//	return rc;
//}
//
//
//void initialize(float *data, unsigned size)
//{
//	for (unsigned i = 0; i < size; ++i)
//		data[i] = .5*(i+1);
//}


int main(void)
{
	std::vector<double> inputPoints;
	ifstream inputFile("input.txt");        // Input file stream object

	// Check if exists and then open the file.
	if (inputFile.good()) {
		// Push items into a vector
		double current_number = 0;
		while (inputFile >> current_number){
			inputPoints.push_back(current_number);
		}

		// Close the file.
		inputFile.close();

		cout << endl;
	}else {
		cout << "Error reading input file!" << endl;
		cout << "Input file must have name \"input.txt\" and must consist of only numbers, no text!" << endl;
		cout << "The program will read the numbers in triples and each 3 numbers will be interpreted as one point." << endl;

		exit(0);
	}

	if(inputPoints.size() % 3 != 0){
		cout << "Number of input numbers should be divisible by 3 (3 dimensions per point)!" << endl;
		cout << "Ignoring last few numbers." << endl;

		int size = inputPoints.size();
		int truncate = size % 3;
		for (int i = 0; i < truncate; ++i) {
			inputPoints.pop_back();
		}
	}

	int nBlocks = 4;
	int nThreads = 32;
	int N = inputPoints.size()/3;
	double dist = 0.5;
	vec3d *points;
	double *distances;
	size_t freeMem, totalMem;
	TopParametric *sur1, *sur2, *sur3, *sur4, *sur5, *sur6;
	BottomParametric *sur7, *sur8, *sur9, *sur10, *sur11, *sur12;

	Model model = Model(
			12.0,
            59.5/180 * M_PI,
            11.4,
            14.4,
            10,
            1.2,
            50.0/180*M_PI,
            7.2,
            16.8,
            3.5,
            1.35,
            -0.2,
            -0.2,
            0.01,
            1.0,
            6.5);

    Model p0 = Model::getPart(model,0),
          p1 = Model::getPart(model,1),
          p2 = Model::getPart(model,2),
          p3 = Model::getPart(model,3),
          p4 = Model::getPart(model,4),
          p5 = Model::getPart(model,5);

	hipMemGetInfo(&freeMem,&totalMem);
	printf("free memory before data load: %d, total memory: %d\n",freeMem,totalMem);

	// Allocate Unified Memory – accessible from CPU or GPU
	CUDA_CHECK_RETURN(hipMallocManaged(&points, N*sizeof(vec3d)));
	CUDA_CHECK_RETURN(hipMallocManaged(&distances, N*sizeof(double)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur1, sizeof(TopParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur2, sizeof(TopParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur3, sizeof(TopParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur4, sizeof(TopParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur5, sizeof(TopParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur6, sizeof(TopParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur7,  sizeof(BottomParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur8,  sizeof(BottomParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur9,  sizeof(BottomParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur10, sizeof(BottomParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur11, sizeof(BottomParametric)));
	CUDA_CHECK_RETURN(hipMallocManaged(&sur12, sizeof(BottomParametric)));

	hipMemGetInfo(&freeMem,&totalMem);
	printf("free memory after data load: %d, total memory: %d\n",freeMem,totalMem);

	*sur1 = p0.getTopParametric();
	*sur2 = p3.getTopParametric();
	*sur3 = p2.getTopParametric();
	*sur4 = p5.getTopParametric();
	*sur5 = p4.getTopParametric();
	*sur6 = p1.getTopParametric();

	*sur7 = p0.getBottomParametric();
	*sur8 = p3.getBottomParametric();
	*sur9 = p2.getBottomParametric();
	*sur10 = p5.getBottomParametric();
	*sur11 = p4.getBottomParametric();
	*sur12 = p1.getBottomParametric();

	srand(1);

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {

        vec3d a = {inputPoints[i*3 + 0],inputPoints[i*3 + 1],inputPoints[i*3 + 2]};

		points[i] = a;
	}

//	*sur = TopParametric(*c,*c,*c,*c,x[0],x[1],x[2],x[3]);
	vec3d h = sur1->at(0,0);
//	printf("dividing height: %f\n",h.z);

	printf("currently working with %d blocks with %d threads each\n",nBlocks,nThreads);

	kernel<<<nBlocks,nThreads>>>(points,distances,N,sur1,sur2,sur3,sur4,sur5,sur6,sur7,sur8,sur9,sur10,sur11,sur12,h.z);
	hipDeviceSynchronize();

	double sum = 0;
	for(int i = 0; i < N; i++){
		sum += distances[i];
	}
	printf("total distance: %lf\n", sum);
	printf("average distance: %lf\n", sum/N);
	printf("number of points: %d\n", N);

	CUDA_CHECK_RETURN(hipFree(points));
	CUDA_CHECK_RETURN(hipFree(distances));
	CUDA_CHECK_RETURN(hipFree(sur1));
	CUDA_CHECK_RETURN(hipFree(sur2));
	CUDA_CHECK_RETURN(hipFree(sur3));
	CUDA_CHECK_RETURN(hipFree(sur4));
	CUDA_CHECK_RETURN(hipFree(sur5));
	CUDA_CHECK_RETURN(hipFree(sur6));
	CUDA_CHECK_RETURN(hipFree(sur7));
	CUDA_CHECK_RETURN(hipFree(sur8));
	CUDA_CHECK_RETURN(hipFree(sur9));
	CUDA_CHECK_RETURN(hipFree(sur10));
	CUDA_CHECK_RETURN(hipFree(sur11));
	CUDA_CHECK_RETURN(hipFree(sur12));
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

